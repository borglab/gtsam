#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2015-2016 Gael Guennebaud <gael.guennebaud@inria.fr>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

// workaround issue between gcc >= 4.7 and cuda 5.5
#if (defined __GNUC__) && (__GNUC__>4 || __GNUC_MINOR__>=7)
  #undef _GLIBCXX_ATOMIC_BUILTINS
  #undef _GLIBCXX_USE_INT128
#endif

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int

#include "main.h"
#include "gpu_common.h"

// Check that dense modules can be properly parsed by nvcc
#include <Eigen/Dense>

// struct Foo{
//   EIGEN_DEVICE_FUNC
//   void operator()(int i, const float* mats, float* vecs) const {
//     using namespace Eigen;
//   //   Matrix3f M(data);
//   //   Vector3f x(data+9);
//   //   Map<Vector3f>(data+9) = M.inverse() * x;
//     Matrix3f M(mats+i/16);
//     Vector3f x(vecs+i*3);
//   //   using std::min;
//   //   using std::sqrt;
//     Map<Vector3f>(vecs+i*3) << x.minCoeff(), 1, 2;// / x.dot(x);//(M.inverse() *  x) / x.x();
//     //x = x*2 + x.y() * x + x * x.maxCoeff() - x / x.sum();
//   }
// };

template<typename T>
struct coeff_wise {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T x1(in+i);
    T x2(in+i+1);
    T x3(in+i+2);
    Map<T> res(out+i*T::MaxSizeAtCompileTime);
    
    res.array() += (in[0] * x1 + x2).array() * x3.array();
  }
};

template<typename T>
struct complex_sqrt {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef typename T::Scalar ComplexType;
    typedef typename T::Scalar::value_type ValueType;
    const int num_special_inputs = 18;
    
    if (i == 0) {
      const ValueType nan = std::numeric_limits<ValueType>::quiet_NaN();
      typedef Eigen::Vector<ComplexType, num_special_inputs> SpecialInputs;
      SpecialInputs special_in;
      special_in.setZero();
      int idx = 0;
      special_in[idx++] = ComplexType(0, 0);
      special_in[idx++] = ComplexType(-0, 0);
      special_in[idx++] = ComplexType(0, -0);
      special_in[idx++] = ComplexType(-0, -0);
      // GCC's fallback sqrt implementation fails for inf inputs.
      // It is called when _GLIBCXX_USE_C99_COMPLEX is false or if
      // clang includes the GCC header (which temporarily disables
      // _GLIBCXX_USE_C99_COMPLEX)
      #if !defined(_GLIBCXX_COMPLEX) || \
        (_GLIBCXX_USE_C99_COMPLEX && !defined(__CLANG_CUDA_WRAPPERS_COMPLEX))
      const ValueType inf = std::numeric_limits<ValueType>::infinity();
      special_in[idx++] = ComplexType(1.0, inf);
      special_in[idx++] = ComplexType(nan, inf);
      special_in[idx++] = ComplexType(1.0, -inf);
      special_in[idx++] = ComplexType(nan, -inf);
      special_in[idx++] = ComplexType(-inf, 1.0);
      special_in[idx++] = ComplexType(inf, 1.0);
      special_in[idx++] = ComplexType(-inf, -1.0);
      special_in[idx++] = ComplexType(inf, -1.0);
      special_in[idx++] = ComplexType(-inf, nan);
      special_in[idx++] = ComplexType(inf, nan);
      #endif
      special_in[idx++] = ComplexType(1.0, nan);
      special_in[idx++] = ComplexType(nan, 1.0);
      special_in[idx++] = ComplexType(nan, -1.0);
      special_in[idx++] = ComplexType(nan, nan);
      
      Map<SpecialInputs> special_out(out);
      special_out = special_in.cwiseSqrt();
    }
    
    T x1(in + i);
    Map<T> res(out + num_special_inputs + i*T::MaxSizeAtCompileTime);
    res = x1.cwiseSqrt();
  }
};

template<typename T>
struct complex_operators {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef typename T::Scalar ComplexType;
    typedef typename T::Scalar::value_type ValueType;
    const int num_scalar_operators = 24;
    const int num_vector_operators = 23;  // no unary + operator.
    int out_idx = i * (num_scalar_operators + num_vector_operators * T::MaxSizeAtCompileTime);
    
    // Scalar operators.
    const ComplexType a = in[i];
    const ComplexType b = in[i + 1];
    
    out[out_idx++] = +a;
    out[out_idx++] = -a;
    
    out[out_idx++] = a + b;
    out[out_idx++] = a + numext::real(b);
    out[out_idx++] = numext::real(a) + b;
    out[out_idx++] = a - b;
    out[out_idx++] = a - numext::real(b);
    out[out_idx++] = numext::real(a) - b;
    out[out_idx++] = a * b;
    out[out_idx++] = a * numext::real(b);
    out[out_idx++] = numext::real(a) * b;
    out[out_idx++] = a / b;
    out[out_idx++] = a / numext::real(b);
    out[out_idx++] = numext::real(a) / b;
    
    out[out_idx] = a; out[out_idx++] += b;
    out[out_idx] = a; out[out_idx++] -= b;
    out[out_idx] = a; out[out_idx++] *= b;
    out[out_idx] = a; out[out_idx++] /= b;
    
    const ComplexType true_value = ComplexType(ValueType(1), ValueType(0));
    const ComplexType false_value = ComplexType(ValueType(0), ValueType(0));
    out[out_idx++] = (a == b ? true_value : false_value);
    out[out_idx++] = (a == numext::real(b) ? true_value : false_value);
    out[out_idx++] = (numext::real(a) == b ? true_value : false_value);
    out[out_idx++] = (a != b ? true_value : false_value);
    out[out_idx++] = (a != numext::real(b) ? true_value : false_value);
    out[out_idx++] = (numext::real(a) != b ? true_value : false_value);
    
    // Vector versions.
    T x1(in + i);
    T x2(in + i + 1);
    const int res_size = T::MaxSizeAtCompileTime * num_scalar_operators;
    const int size = T::MaxSizeAtCompileTime;
    int block_idx = 0;
    
    Map<VectorX<ComplexType>> res(out + out_idx, res_size);
    res.segment(block_idx, size) = -x1;
    block_idx += size;
    
    res.segment(block_idx, size) = x1 + x2;
    block_idx += size;
    res.segment(block_idx, size) = x1 + x2.real();
    block_idx += size;
    res.segment(block_idx, size) = x1.real() + x2;
    block_idx += size;
    res.segment(block_idx, size) = x1 - x2;
    block_idx += size;
    res.segment(block_idx, size) = x1 - x2.real();
    block_idx += size;
    res.segment(block_idx, size) = x1.real() - x2;
    block_idx += size;
    res.segment(block_idx, size) = x1.array() * x2.array();
    block_idx += size;
    res.segment(block_idx, size) = x1.array() * x2.real().array();
    block_idx += size;
    res.segment(block_idx, size) = x1.real().array() * x2.array();
    block_idx += size;
    res.segment(block_idx, size) = x1.array() / x2.array();
    block_idx += size;
    res.segment(block_idx, size) = x1.array() / x2.real().array();
    block_idx += size;
    res.segment(block_idx, size) = x1.real().array() / x2.array();
    block_idx += size;
    
    res.segment(block_idx, size) = x1; res.segment(block_idx, size) += x2;
    block_idx += size;
    res.segment(block_idx, size) = x1; res.segment(block_idx, size) -= x2;
    block_idx += size;
    res.segment(block_idx, size) = x1; res.segment(block_idx, size).array() *= x2.array();
    block_idx += size;
    res.segment(block_idx, size) = x1; res.segment(block_idx, size).array() /= x2.array();
    block_idx += size;

    const T true_vector = T::Constant(true_value);
    const T false_vector = T::Constant(false_value);
    res.segment(block_idx, size) = (x1 == x2 ? true_vector : false_vector);
    block_idx += size;
    // Mixing types in equality comparison does not work.
    // res.segment(block_idx, size) = (x1 == x2.real() ? true_vector : false_vector);
    // block_idx += size;
    // res.segment(block_idx, size) = (x1.real() == x2 ? true_vector : false_vector);
    // block_idx += size;
    res.segment(block_idx, size) = (x1 != x2 ? true_vector : false_vector);
    block_idx += size;
    // res.segment(block_idx, size) = (x1 != x2.real() ? true_vector : false_vector);
    // block_idx += size;
    // res.segment(block_idx, size) = (x1.real() != x2 ? true_vector : false_vector);
    // block_idx += size;
  }
};

template<typename T>
struct replicate {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T x1(in+i);
    int step   = x1.size() * 4;
    int stride = 3 * step;
    
    typedef Map<Array<typename T::Scalar,Dynamic,Dynamic> > MapType;
    MapType(out+i*stride+0*step, x1.rows()*2, x1.cols()*2) = x1.replicate(2,2);
    MapType(out+i*stride+1*step, x1.rows()*3, x1.cols()) = in[i] * x1.colwise().replicate(3);
    MapType(out+i*stride+2*step, x1.rows(), x1.cols()*3) = in[i] * x1.rowwise().replicate(3);
  }
};

template<typename T>
struct alloc_new_delete {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    int offset = 2*i*T::MaxSizeAtCompileTime;
    T* x = new T(in + offset);
    Eigen::Map<T> u(out + offset);
    u = *x;
    delete x;
    
    offset += T::MaxSizeAtCompileTime;
    T* y = new T[1];
    y[0] = T(in + offset);
    Eigen::Map<T> v(out + offset);
    v = y[0];    
    delete[] y;
  }
};

template<typename T>
struct redux {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    int N = 10;
    T x1(in+i);
    out[i*N+0] = x1.minCoeff();
    out[i*N+1] = x1.maxCoeff();
    out[i*N+2] = x1.sum();
    out[i*N+3] = x1.prod();
    out[i*N+4] = x1.matrix().squaredNorm();
    out[i*N+5] = x1.matrix().norm();
    out[i*N+6] = x1.colwise().sum().maxCoeff();
    out[i*N+7] = x1.rowwise().maxCoeff().sum();
    out[i*N+8] = x1.matrix().colwise().squaredNorm().sum();
  }
};

template<typename T1, typename T2>
struct prod_test {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T1::Scalar* in, typename T1::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T1::Scalar, T1::RowsAtCompileTime, T2::ColsAtCompileTime> T3;
    T1 x1(in+i);
    T2 x2(in+i+1);
    Map<T3> res(out+i*T3::MaxSizeAtCompileTime);
    res += in[i] * x1 * x2;
  }
};

template<typename T1, typename T2>
struct diagonal {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T1::Scalar* in, typename T1::Scalar* out) const
  {
    using namespace Eigen;
    T1 x1(in+i);
    Map<T2> res(out+i*T2::MaxSizeAtCompileTime);
    res += x1.diagonal();
  }
};

template<typename T>
struct eigenvalues_direct {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T::Scalar, T::RowsAtCompileTime, 1> Vec;
    T M(in+i);
    Map<Vec> res(out+i*Vec::MaxSizeAtCompileTime);
    T A = M*M.adjoint();
    SelfAdjointEigenSolver<T> eig;
    eig.computeDirect(A);
    res = eig.eigenvalues();
  }
};

template<typename T>
struct eigenvalues {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    typedef Matrix<typename T::Scalar, T::RowsAtCompileTime, 1> Vec;
    T M(in+i);
    Map<Vec> res(out+i*Vec::MaxSizeAtCompileTime);
    T A = M*M.adjoint();
    SelfAdjointEigenSolver<T> eig;
    eig.compute(A);
    res = eig.eigenvalues();
  }
};

template<typename T>
struct matrix_inverse {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    using namespace Eigen;
    T M(in+i);
    Map<T> res(out+i*T::MaxSizeAtCompileTime);
    res = M.inverse();
  }
};

template<typename T>
struct numeric_limits_test {
  EIGEN_DEVICE_FUNC
  void operator()(int i, const typename T::Scalar* in, typename T::Scalar* out) const
  {
    EIGEN_UNUSED_VARIABLE(in)
    int out_idx = i * 5;
    out[out_idx++] = numext::numeric_limits<float>::epsilon();
    out[out_idx++] = (numext::numeric_limits<float>::max)();
    out[out_idx++] = (numext::numeric_limits<float>::min)();
    out[out_idx++] = numext::numeric_limits<float>::infinity();
    out[out_idx++] = numext::numeric_limits<float>::quiet_NaN();
  }
};

template<typename Type1, typename Type2>
bool verifyIsApproxWithInfsNans(const Type1& a, const Type2& b, typename Type1::Scalar* = 0) // Enabled for Eigen's type only
{
  if (a.rows() != b.rows()) {
    return false;
  }
  if (a.cols() != b.cols()) {
    return false;
  }
  for (Index r = 0; r < a.rows(); ++r) {
    for (Index c = 0; c < a.cols(); ++c) {
      if (a(r, c) != b(r, c)
          && !((numext::isnan)(a(r, c)) && (numext::isnan)(b(r, c))) 
          && !test_isApprox(a(r, c), b(r, c))) {
        return false;
      }
    }
  }
  return true;
}

template<typename Kernel, typename Input, typename Output>
void test_with_infs_nans(const Kernel& ker, int n, const Input& in, Output& out)
{
  Output out_ref, out_gpu;
  #if !defined(EIGEN_GPU_COMPILE_PHASE)
  out_ref = out_gpu = out;
  #else
  EIGEN_UNUSED_VARIABLE(in);
  EIGEN_UNUSED_VARIABLE(out);
  #endif
  run_on_cpu (ker, n, in,  out_ref);
  run_on_gpu(ker, n, in, out_gpu);
  #if !defined(EIGEN_GPU_COMPILE_PHASE)
  verifyIsApproxWithInfsNans(out_ref, out_gpu);
  #endif
}

EIGEN_DECLARE_TEST(gpu_basic)
{
  ei_test_init_gpu();
  
  int nthreads = 100;
  Eigen::VectorXf in, out;
  Eigen::VectorXcf cfin, cfout;
  
  #if !defined(EIGEN_GPU_COMPILE_PHASE)
  int data_size = nthreads * 512;
  in.setRandom(data_size);
  out.setConstant(data_size, -1);
  cfin.setRandom(data_size);
  cfout.setConstant(data_size, -1);
  #endif
  
  CALL_SUBTEST( run_and_compare_to_gpu(coeff_wise<Vector3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(coeff_wise<Array44f>(), nthreads, in, out) );

#if !defined(EIGEN_USE_HIP)
  // FIXME
  // These subtests result in a compile failure on the HIP platform
  //
  //  eigen-upstream/Eigen/src/Core/Replicate.h:61:65: error:
  //           base class 'internal::dense_xpr_base<Replicate<Array<float, 4, 1, 0, 4, 1>, -1, -1> >::type'
  //           (aka 'ArrayBase<Eigen::Replicate<Eigen::Array<float, 4, 1, 0, 4, 1>, -1, -1> >') has protected default constructor
  CALL_SUBTEST( run_and_compare_to_gpu(replicate<Array4f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(replicate<Array33f>(), nthreads, in, out) );

  // HIP does not support new/delete on device.
  CALL_SUBTEST( run_and_compare_to_gpu(alloc_new_delete<Vector3f>(), nthreads, in, out) );
#endif
  
  CALL_SUBTEST( run_and_compare_to_gpu(redux<Array4f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(redux<Matrix3f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(prod_test<Matrix3f,Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(prod_test<Matrix4f,Vector4f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(diagonal<Matrix3f,Vector3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(diagonal<Matrix4f,Vector4f>(), nthreads, in, out) );

  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix2f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(matrix_inverse<Matrix4f>(), nthreads, in, out) );
  
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues_direct<Matrix3f>(), nthreads, in, out) );
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues_direct<Matrix2f>(), nthreads, in, out) );

  // Test std::complex.
  CALL_SUBTEST( run_and_compare_to_gpu(complex_operators<Vector3cf>(), nthreads, cfin, cfout) );
  CALL_SUBTEST( test_with_infs_nans(complex_sqrt<Vector3cf>(), nthreads, cfin, cfout) );

  // numeric_limits
  CALL_SUBTEST( test_with_infs_nans(numeric_limits_test<Vector3f>(), 1, in, out) );

#if defined(__NVCC__)
  // FIXME
  // These subtests compiles only with nvcc and fail with HIPCC and clang-cuda
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues<Matrix4f>(), nthreads, in, out) );
  typedef Matrix<float,6,6> Matrix6f;
  CALL_SUBTEST( run_and_compare_to_gpu(eigenvalues<Matrix6f>(), nthreads, in, out) );
#endif
}
